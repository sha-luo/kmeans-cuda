
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c){
    *c = *a + *b;
}

int main(void){
    int a, b, c;
    int *d_a, *d_b, *d_c;
    int size = sizeof(int);

    a = 3;
    b = 8;

    // allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // copy inputs to device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    // Launch add() kernal on GPU
    add<<<1,1>>>(d_a, d_b, d_c);

    //copy result back to host
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    // print result
    printf("The result of %d + %d is %d\n", a, b, c);

    //clean up
    hipFree(d_a); 
    hipFree(d_b); 
    hipFree(d_c);
    return 0;
}